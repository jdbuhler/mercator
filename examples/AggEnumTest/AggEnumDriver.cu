#include "hip/hip_runtime.h"
//
// AGGENUMDRIVER.CU
// Aggregation and Enumeration functionality test
//
// MERCATOR
// Copyright (C) 2018 Washington University in St. Louis; all rights reserved.
//

#include <iostream>
#include <cstdlib>

#include "AggEnum.cuh"

using namespace std;

int main()
{
 // const unsigned int NVALUES = 1000000000; // one BEEEELLION values
  //const unsigned int NVALUES = 1000000;	//DID MAIN TEST WITH THIS ONE <---------------------------------
  //const unsigned int NVALUES = 4000; // one BEEEELLION values

  ////const unsigned int NVALUES = 800000; // one BEEEELLION values

  ////const unsigned int NVALUES = 1000000; // CURRENT BREAKING TEST
  //const unsigned int NVALUES = 74000; // one BEEEELLION values
  //const unsigned int NVALUES = 7400; // one BEEEELLION values
  //const unsigned int NVALUES = 256; // one BEEEELLION values
  //const unsigned int NVALUES = 220000; // one BEEEELLION values
  //const unsigned int NVALUES = 10000000; // one BEEEELLION values
  //const unsigned int NVALUES = 35840; // one BEEEELLION values
  //const unsigned int NVALUES = 143360; // one BEEEELLION values
  //const unsigned int NVALUES = 107520; // one BEEEELLION values

  //const unsigned int NVALUES = 71934;
  //const unsigned int NVALUES = 71935;

  //const unsigned int NVALUES = 129024;
  //const unsigned int NVALUES = 129025;

  //const unsigned int NVALUES = 1000;
  //const unsigned int NVALUES = 514;
  //const unsigned int NVALUES = 20;
  //const unsigned int NVALUES = 10;

  //EVEN TESTS 2B
  //const unsigned int NVALUES = 64000000;
  //const unsigned int NVALUES = 16000000;
  //const unsigned int NVALUES = 4000000;
  //const unsigned int NVALUES = 1000000;
  
  //const unsigned int MULTIPLIER = 32;
  //const unsigned int MULTIPLIER = 128;
  //const unsigned int MULTIPLIER = 512;
  //const unsigned int MULTIPLIER = 2048;
  
  //ODD TESTS 2B
  //const unsigned int NVALUES = 15875969;
  //const unsigned int NVALUES = 10666667;


  ///////////////
  //EVEN TESTS 1B
  //const unsigned int NVALUES = 32000000;
  //const unsigned int NVALUES = 8000000;
  //const unsigned int NVALUES = 2000000;
  //const unsigned int NVALUES = 500000;
  
  //const unsigned int MULTIPLIER = 32;
  //const unsigned int MULTIPLIER = 128;
  //const unsigned int MULTIPLIER = 512;
  //const unsigned int MULTIPLIER = 2048;


  ///////////////
  //ODD TESTS 1B
  //const unsigned int NVALUES = 16000000;
  //const unsigned int NVALUES = 5333333;
  //const unsigned int NVALUES = 1333333;
  //const unsigned int NVALUES = 1000000;
  
  //const unsigned int MULTIPLIER = 64;
  //const unsigned int MULTIPLIER = 192;
  //const unsigned int MULTIPLIER = 768;
  //const unsigned int MULTIPLIER = 1024;


  ///////////////
  //ODD TESTS2 1B
  //const unsigned int NVALUES = 7937985;
  //const unsigned int NVALUES = 1996101;
  //const unsigned int NVALUES = 999024;
  const unsigned int NVALUES = 499756;
  
  //const unsigned int MULTIPLIER = 129;
  //const unsigned int MULTIPLIER = 513;
  //const unsigned int MULTIPLIER = 1025;
  const unsigned int MULTIPLIER = 2049;


  //const unsigned int NVALUES = 500000;
  //const unsigned int MULTIPLIER = 2048;


  //const unsigned int NVALUES = 32000000;
  //const unsigned int MULTIPLIER = 32;

  //cout << "HERE1" << endl;
  srand(0);
  //cout << "HERE2" << endl;
  
  unsigned int *inputValues = new unsigned int [NVALUES];
  unsigned int *outputValues = new unsigned int [NVALUES*MULTIPLIER];
  //cout << "HERE3" << endl;
  
  unsigned int total = 0;
  for (unsigned int j = 0; j < NVALUES; j++) {
    //inputValues[j] = rand();

    //ALT TEST
    inputValues[j] = MULTIPLIER;
    total += MULTIPLIER;


    //MAIN TEST
    //inputValues[j] = j % (MULTIPLIER + 1);
    //total += j % (MULTIPLIER + 1);
  }
  //cout << "HERE4" << endl;
  
  // begin MERCATOR usage
  
  Mercator::Buffer<unsigned int> inputBuffer(NVALUES);
  //Mercator::Buffer<unsigned int> outputBufferAccept(NVALUES);
  Mercator::Buffer<unsigned int> outputBufferAccept(NVALUES*MULTIPLIER);
  //cout << "HERE5" << endl;
 
  //int x;
  //cin >> x; 
  AggEnum efapp;
  
  //cout << "HERE6" << endl;
  efapp.src.setSource(inputBuffer);
  efapp.snk.setSink(outputBufferAccept);
  //cout << "HERE7" << endl;
  
  // move data into the input buffer
  inputBuffer.set(inputValues, NVALUES);
  //cout << "HERE8" << endl;
  
  cout << "RUNNING APP. . . " << endl;
  //unsigned int max = UINT_MAX * 32;
  //cout << UINT_MAX << "\t\t" << max << endl;
  efapp.run();
  cout << "APP FINISHED. . ." << endl;
  
  // get data out of the output buffer
  unsigned int outSize = outputBufferAccept.size();
  outputBufferAccept.get(outputValues, outSize);
  
  // end MERCATOR usage
  
  cout << "# outputs = " << outSize << endl;
  cout << "# expected outputs = " << total << endl;

  //for(unsigned int j = 0; j < NVALUES*MULTIPLIER; j+=1)
	//cout << "out[" << j << "]:\t" << outputValues[j] << endl;
  
  delete [] inputValues;
  delete [] outputValues;
  
  return 0;
}
