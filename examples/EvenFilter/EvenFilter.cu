#include "hip/hip_runtime.h"
//
// EVENFILER.CU
// Device-side app to filter even numbers
//
// MERCATOR
// Copyright (C) 2018 Washington University in St. Louis; all rights reserved.
//

#include "EvenFilter_dev.cuh"
#define UPPERBOUNd 750000
#define LOWERBOUND 700000 


__device__
unsigned int munge(unsigned int key)
{
  key = (key << 15) - key - 1;
  key = key ^ (key >> 12);
  key = key + (key << 2);
  key = key ^ (key >> 4);
  key = key * 2057;
  key = key ^ (key >> 16);
  return key;
}

__device__
void EvenFilter_dev::
Filter::init()
{
#ifdef INSTRUMENT_FG_TIME
 //set upperbound for data collection
  if(IS_BOSS()){
    setFGContainerBounds((unsigned long long)LOWERBOUND, (unsigned long long)UPPERBOUNd);
    }
  __syncthreads(); // all threads must see updates to the bounds
#endif  
}

//
// Hash each input item and return only those hash values that
// are even numbers.
//
__device__
void EvenFilter_dev::
Filter::run(const unsigned int& inputItem, InstTagT nodeIdx)
{
  
  unsigned int v = munge(inputItem);
  for (int i=0; i<10000; i++){
    v = munge(v);
  }
  // If no channel is specified, push sends a value to the module's
  // first output channel.
  if (v % 2 == 0)
    push(v, nodeIdx); // eqv to "push(v, nodeIdx, Out::accept);"
}
__device__
void EvenFilter_dev::
Filter::cleanup()
{
}
