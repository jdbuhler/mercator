#include "hip/hip_runtime.h"
//
// SYNTHGAINS.CU
// Device-side app to test different streaming behavior.
//
// MERCATOR
// Copyright (C) 2021 Washington University in St. Louis; all rights reserved.
//

#include "SynthGain_dev.cuh"

//Set the number of iterations of extra work for each input.
#define ITERS 1000

//Function for doing extra work on every output.
__device__
size_t extra_work(size_t key)
{
  for(unsigned int i = 0; i < ITERS; ++i) {
  key = (key << 15) - key - 1;
  key = key ^ (key >> 12);
  key = key + (key << 2);
  key = key ^ (key >> 4);
  key = key * 2057;
  key = key ^ (key >> 16);
  }
  return key;
}

__MDECL__
void SynthGain_dev::
type1<InputView>::run(const size_t& inputItem, unsigned int nInputs)
{
  unsigned int tid = threadIdx.x;

  auto params = getParams();
  float g = params->avgGain;
  unsigned int fullG = (unsigned int)(g);
  float partG = g - fullG;

  unsigned int totalOut = 0;
  
  //Determine which threads should produce outputs.
  if (tid < nInputs) {
    totalOut = fullG;

    //Determine which threads should produce an extra output, when there is a partial gain.
    if (partG > 0.0) {
      totalOut += ((unsigned int)(nInputs * partG) <= tid ? 0 : 1);
    }
  }

  using R = Mercator::BlockReduce<unsigned int, THREADS_PER_BLOCK>;
  auto &bcast = Mercator::broadcast<unsigned int, THREADS_PER_BLOCK>;
  
  unsigned int blockMaxHits = R::max(totalOut);
  blockMaxHits = bcast(blockMaxHits, 0);

  //DEBUG
  //printf("[%d, %d] g=%lf\tfullG=%d\tpartG=%lf\ttotalOut=%d\tblockMaxHits=%d\tnInputs*partG=%d\tpartTest=%d\n", blockIdx.x, tid, g, fullG, partG, totalOut, blockMaxHits, (unsigned int)(nInputs*partG), ((unsigned int)(nInputs * partG) <= tid ? 0 : 1));

  //Output the total number of items for this thread.  Do extra work on the input before outputing.
  for(unsigned int i = 0; i < blockMaxHits; ++i)
    push(extra_work(inputItem), i < totalOut); // defaults to pushing to Out::accept

}

