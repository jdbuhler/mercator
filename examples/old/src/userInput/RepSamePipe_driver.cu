#ifndef __REPSAMEPIPE_DRIVER_CU
#define __REPSAMEPIPE_DRIVER_CU


/**
 * @brief Driver (test harnesses) for Mercator app
 *          PipeSameType.
 */

#include <iostream>
#include <cstdlib>

#include "support/util.cuh"

#include "driver_config.cuh"

#include "./tests/PipeEltT.cuh"   
#include "./tests/NodeDataT.cuh" 

#include "./utils.cuh"

// only support 1-to-1 mapping for now
#if MAPPING_1TO1
  #include "../codegenInput/RepSamePipe_1to1map.cuh"
  #define APP_TYPE RepSamePipe_1to1map
//#elif MAPPING_1TO2
//  #include "../codegenInput/SameTypePipe_1to2map.cuh"
//  #define APP_TYPE SameTypePipe_1to2map
//#elif MAPPING_1TO4
//  #include "../codegenInput/SameTypePipe_1to4map.cuh"
//  #define APP_TYPE SameTypePipe_1to4map
//#elif MAPPING_2TO1
//  #include "../codegenInput/SameTypePipe_2to1map.cuh"
//  #define APP_TYPE SameTypePipe_2to1map
//#elif MAPPING_4TO1
//  #include "../codegenInput/SameTypePipe_4to1map.cuh"
//  #define APP_TYPE SameTypePipe_4to1map
#else
  #error "INVALID MAPPING SELECTION"
#endif

#define PRINT_INPUT_BUFFER_REPSAMEPIPE 0
#define PRINT_OUTPUT_BUFFERS_REPSAMEPIPE 0

void run_repSamePipe()
{
  // replication factor from inputs to outputs-- should equal number
  //   of parallel pipes
//  constexpr int GAMMA = 2;
  constexpr int GAMMA = 1;

  const int OUT_BUFFER_CAPACITY = GAMMA * IN_BUFFER_CAPACITY; 

  // set input info
  constexpr int NUM_INPUTS = IN_BUFFER_CAPACITY;

#if 1
  // print experiment params if desired
  // NB: all possible topos included for sanity check
  // convert topology indicators to string
#if RUN_SAMETYPEPIPE
      const char topoString[] = "SameTypePipe";
#elif RUN_DIFFTYPEPIPE
      const char topoString[] = "DiffTypePipe";
#elif RUN_UBERNODEPIPE
      const char topoString[] = "UberNodePipe";
#elif RUN_SELFLOOPPIPE
      const char topoString[] = "SelfLoop";
#elif RUN_REPSAMEPIPE
      const char topoString[] = "RepSamePipe";
#elif RUN_REPDIFFPIPE
      const char topoString[] = "RepDiffPipe";
#else
      const char topoString[] = "NONSTANDARD";
#endif

      // convert mapping indicators to string
#if MAPPING_1TO1
      const char mapString[] = "1-to-1";
#elif MAPPING_1TO2
      const char mapString[] = "1-to-2";
#elif MAPPING_1TO4
      const char mapString[] = "1-to-4";
#elif MAPPING_2TO1
      const char mapString[] = "2-to-1";
#elif MAPPING_4TO1
      const char mapString[] = "4-to-1";
#else
      const char mapString[] = "NONSTANDARD";
#endif

      // print app metadata
      printf("APP PARAMS: TOPOLOGY: %s ELTS-TO-THREADS MAPPING: %s FILTER_RATE: %.2f WORK_ITERS: %d INPUTS: %d\n", 
          topoString, mapString, FILTER_RATE, WORK_ITERS, NUM_INPUTS); 

      //debug
//      printf("Size of input item: %d\n", sizeof(PipeEltT));
#endif


  int inputSeed;
#if USE_REPEATABLE_INPUTS
  inputSeed = 1919;
#else
  inputsSeed = time(0);
#endif

  // alloc input data
  PipeEltT* inBufferData;
  hipMallocManaged(&inBufferData, IN_BUFFER_CAPACITY * sizeof(PipeEltT));
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );

#if PRINT_MEM_USAGE
  printf("Allocating global mem for input buffer, capacity: %d size (bytes): %ld\n", IN_BUFFER_CAPACITY, (long long)(IN_BUFFER_CAPACITY * sizeof(PipeEltT)));
#endif

  // create buffers
  Mercator::InputBuffer<PipeEltT>* inBuffer = new Mercator::InputBuffer<PipeEltT>(inBufferData, IN_BUFFER_CAPACITY);

  Mercator::OutputBuffer<PipeEltT>* outBufferAccept = new Mercator::OutputBuffer<PipeEltT>(OUT_BUFFER_CAPACITY);

  // fill input buffer
  srand(inputSeed);
  for(int i=0; i < IN_BUFFER_CAPACITY; ++i)
  {
    int nextID = rand() % IN_BUFFER_CAPACITY;
    // set random TTL in range [0,9]
    int nextTTL = rand() % 10;
    int loopCount = 0;
    int numWorkIters = WORK_ITERS; 

    inBuffer->add(PipeEltT(nextID, loopCount, numWorkIters)); 
  }

  // copy of input data for later validation
  PipeEltT* inBufferData_gold = new PipeEltT[IN_BUFFER_CAPACITY];
  for(int i=0; i < IN_BUFFER_CAPACITY; ++i)
  {
    inBufferData_gold[i] = inBuffer->peek(i);
  }


#if PRINT_INPUT_BUFFER_REPSAMEPIPE
  // print input buffer contents
  printf("RepSamePipe, InputBuffer (%p):\n", inBuffer);
  for(int i=0; i < IN_BUFFER_CAPACITY; ++i)
  {
    printf("[%d]: ID: %d loop count: %d work loops: %d Int result: %d Double result: %lf Float result: %f\n", 
        i, inBuffer->peek(i).get_ID(),
        inBuffer->peek(i).get_loopCount(),
        inBuffer->peek(i).get_workIters(),
        inBuffer->peek(i).get_intResult(),
        inBuffer->peek(i).get_doubleResult(),
        inBuffer->peek(i).get_floatResult());
  }
#endif

  // create app object
  APP_TYPE* repSamePipe = new APP_TYPE();

  // set up each main node in pipeline
  constexpr int NUM_NODES = 5;

  NodeDataT* myNodeData[NUM_NODES];

  int upperBd = NUM_INPUTS;
  int lastUpperBd = upperBd;  // final (lowest) filter value; used for
                              //  validation
  for(int i=0; i < NUM_NODES; ++i)
  {
    // debug
//    printf("Setting upper bound of node %d to %d\n", i, upperBd);
    if(i == NUM_NODES - 1)
      lastUpperBd = upperBd;
    // end debug

    myNodeData[i] = new NodeDataT(float(FILTER_RATE), upperBd);
    upperBd -= (int)(FILTER_RATE * (float)upperBd);
  }

  repSamePipe->A->set_nodeUserData(APP_TYPE::A::Node::A11node, myNodeData[0]);
  repSamePipe->A->set_nodeUserData(APP_TYPE::A::Node::A12node, myNodeData[1]);
  repSamePipe->A->set_nodeUserData(APP_TYPE::A::Node::A13node, myNodeData[2]);
  repSamePipe->A->set_nodeUserData(APP_TYPE::A::Node::A14node, myNodeData[3]);
  repSamePipe->A->set_nodeUserData(APP_TYPE::A::Node::A15node, myNodeData[4]);
  repSamePipe->A->set_nodeUserData(APP_TYPE::A::Node::A21node, myNodeData[0]);
  repSamePipe->A->set_nodeUserData(APP_TYPE::A::Node::A22node, myNodeData[1]);
  repSamePipe->A->set_nodeUserData(APP_TYPE::A::Node::A23node, myNodeData[2]);
  repSamePipe->A->set_nodeUserData(APP_TYPE::A::Node::A24node, myNodeData[3]);
  repSamePipe->A->set_nodeUserData(APP_TYPE::A::Node::A25node, myNodeData[4]);

  // associate buffers with nodes
  repSamePipe->sourceNode->set_inBuffer(inBuffer);
  repSamePipe->sinkNodeAccept1->set_outBuffer(outBufferAccept);
  repSamePipe->sinkNodeAccept2->set_outBuffer(outBufferAccept);

  // run main function
  repSamePipe->run();

  std::cout << "REP-SAME-PIPE APP FINISHED.\n" ;

  /////////////////// output processing

  PipeEltT* outDataAccept = outBufferAccept->get_data();

  // print contents of output buffer
#if PRINT_OUTPUT_BUFFERS_REPSAMEPIPE
  std::cout << " Output buffer: \n" ;

  printf("Rep-same-pipe, OutBufferAccept (%p):\n", outBufferAccept);
  for(int i=0; i < outBufferAccept->size(); ++i)
    printf("[%d]: ID: %d work loops: %d Int result: %d Double result: %lf Float result: %f\n", 
        i, outDataAccept[i].get_ID(), 
        outDataAccept[i].get_workIters(),
        outDataAccept[i].get_intResult(),
        outDataAccept[i].get_doubleResult(),
        outDataAccept[i].get_floatResult());

#endif   // print contents of output buffer

  //////////////
  // validate output against pristine copy of input buffer
  // NB: since pipeline is replicated, TWO copies of every valid output
  //     should exist in output buffer
  //////////////

  bool allValid = validate_lowpassFilterApp_outputs(
    inBufferData_gold,
    IN_BUFFER_CAPACITY,
    outDataAccept,
    outBufferAccept->size(),
    (1-FILTER_RATE) * lastUpperBd,
    GAMMA);

  if(allValid)
    printf("OUTPUT VALIDATED CORRECT, %d items.\n", outBufferAccept->size());
  else
    printf("OUTPUT CONTAINS ERRORS.\n");
  /////////////////////////////////////////
    


  // cleanup
  for(int i=0; i < NUM_NODES; ++i)
  {
    hipFree(myNodeData[i]);
    gpuErrchk( hipPeekAtLastError() );
  }

  hipFree(inBufferData);
  gpuErrchk( hipPeekAtLastError() );
  hipFree(repSamePipe);
  gpuErrchk( hipPeekAtLastError() );

  delete[] inBufferData_gold;
}

#endif
