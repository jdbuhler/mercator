#ifndef __SYNTHAPP2_DRIVER_CU
#define __SYNTHAPP2_DRIVER_CU


/**
 * @brief Driver (test harnesses) for Mercator app
 *          SynthApp2.
 */

#include <iostream>

#include "driver_config.cuh"

#include "./tests/datatypes.h"
#include "../codegenInput/SynthApp2.cuh"

#define PRINT_OUTPUT_BUFFERS 1

void run_synthApp2()
{
  // output buffers accommodates 4 outputs/input
  const int OUT_BUFFER_CAPACITY1 = 4 * IN_BUFFER_CAPACITY; 
  const int OUT_BUFFER_CAPACITY2 = 16 * IN_BUFFER_CAPACITY; 

  int* inBufferData;
  hipMallocManaged(&inBufferData, IN_BUFFER_CAPACITY * sizeof(int));
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );

#if PRINT_MEM_USAGE
  printf("Allocating global mem for input buffer, capacity: %d size (bytes): %ld\n", IN_BUFFER_CAPACITY, (long long)(IN_BUFFER_CAPACITY * sizeof(int)));
#endif

  Mercator::InputBuffer<int>* inBuffer = new Mercator::InputBuffer<int>(inBufferData, IN_BUFFER_CAPACITY);


  Mercator::OutputBuffer<int>* outBuffer1 = new Mercator::OutputBuffer<int>(OUT_BUFFER_CAPACITY1);
  Mercator::OutputBuffer<int>* outBuffer2 = new Mercator::OutputBuffer<int>(OUT_BUFFER_CAPACITY2);
  Mercator::OutputBuffer<int>* outBuffer3 = new Mercator::OutputBuffer<int>(OUT_BUFFER_CAPACITY2);

  // fill input buffer
  for(int i=0; i < IN_BUFFER_CAPACITY; ++i)
  {
    inBuffer->add(i); 
  }

  // create app object
  SynthApp2* synthApp2 = new SynthApp2();

  // set node-, module-, app-level user data
  MyModuleData* filter1data = new MyModuleData(2);
  synthApp2->Filter1->set_userData(filter1data); // regular module

  //  MyModuleData* filter2data = new MyModuleData(7*TPI);
  MyModuleData* filter2data = new MyModuleData(7);
  synthApp2->Filter2->set_userData(filter2data); // regular module

  MyNodeData* filter1nodeData = new MyNodeData(3);
  
  synthApp2->Filter1->set_nodeUserData(SynthApp2::Filter1::Node::filter1node, filter1nodeData);

  MyNodeData* filter2nodeData = new MyNodeData(5);
  
  synthApp2->Filter2->set_nodeUserData(SynthApp2::Filter2::Node::filter2node, filter2nodeData);

  // associate buffers with nodes
  synthApp2->sourceNode->set_inBuffer(inBuffer);
  synthApp2->sinkNodeReject1->set_outBuffer(outBuffer1);
  synthApp2->sinkNodeReject2->set_outBuffer(outBuffer2);
  synthApp2->sinkNodeAccept->set_outBuffer(outBuffer3);

  // run main function
  synthApp2->run();

  std::cout << "SynthApp2 finished.\n" ;

  // print contents of output buffer
#if PRINT_OUTPUT_BUFFERS
  std::cout << " Output buffers: \n" ;

  int* outData1 = outBuffer1->get_data();
  printf("SynthApp2, OutBuffer1 (%p):\n", outBuffer1);
  for(int i=0; i < outBuffer1->size(); ++i)
    printf("[%d]: %d\n", i, outData1[i]);

  int* outData2 = outBuffer2->get_data();
  printf("SynthApp2, OutBuffer2 (%p):\n", outBuffer2);
  for(int i=0; i < outBuffer2->size(); ++i)
    printf("[%d]: %d\n", i, outData2[i]);

  int* outData3 = outBuffer3->get_data();
  printf("SynthApp2, OutBuffer3 (%p):\n", outBuffer3);
  for(int i=0; i < outBuffer3->size(); ++i)
    printf("[%d]: %d\n", i, outData3[i]);
#endif   // print contents of output buffer

  // cleanup
  hipFree(inBufferData);
  hipFree(synthApp2);

}

#endif
