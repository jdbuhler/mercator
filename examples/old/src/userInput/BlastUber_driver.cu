#include "hip/hip_runtime.h"
#ifndef __BLASTUBER_DRIVER_CU
#define __BLASTUBER_DRIVER_CU

/**
 * @brief Driver (test harnesses) for Mercator app
 *          BlastApp.
 */

#include <iostream>
#include <string>
#include <fstream>
#include <math.h>
#include <limits.h>
#include <vector>

#include "./tests/datatypes.h"
#include "./tests/blastData.cuh"

#include "driver_config.cuh"

#if MAPPING_1TO1
  #include "../codegenInput/BlastUberApp_1to1map.cuh"
  #define APP_TYPE BlastUberApp_1to1map
#elif MAPPING_1TO2
  #include "../codegenInput/BlastUberApp_1to2map.cuh"
  #define APP_TYPE BlastUberApp_1to2map
#elif MAPPING_1TO4
  #include "../codegenInput/BlastUberApp_1to4map.cuh"
  #define APP_TYPE BlastUberApp_1to4map
#elif MAPPING_2TO1
  #include "../codegenInput/BlastUberApp_2to1map.cuh"
  #define APP_TYPE BlastUberApp_2to1map
#elif MAPPING_4TO1
  #include "../codegenInput/BlastUberApp_4to1map.cuh"
  #define APP_TYPE BlastUberApp_4to1map
#else
  #error "INVALID MAPPING SELECTION"
#endif

// BLAST-specific param
#define MAX_DIFF 128

#define PRINT_OUTPUT_BUFFERS_BLAST 0

#if 0
struct node {
	int x;
	node* next;
};
#endif

void run_blastUberApp()
{
	const int HASH_SIZE = 65536;
  // set up input buffer
  //const int BUFFER_CAPACITY = 1024;

	//Read database and query files
        // NB: Query file should be in text format with chars 'ACGT' only.
        //     DB file should be in packed format, with 2 bits per base (4 chars per byte), 
        //       using the following encoding: 00 = 'A', 01 = 'C',
        //       10 = 'G', 11 = 'T'

//	std::string queryFilename = "./bin/BlastData/query.txt"; // NB: this one works!
//	std::string dbFilename = "./bin/BlastData/d2.txt";

//	std::string queryFilename = "./bin/BlastData/query-replicated.txt"; // NB: this one works!
//	std::string dbFilename = "./bin/BlastData/d2-replicated.txt";

//	std::string queryFilename = "./bin/BlastData/salmonella.txt";
//	std::string queryFilename = "./bin/BlastData/salmonella-5k.txt";
//	std::string queryFilename = "./bin/BlastData/salmonella-2k.txt";
//	std::string queryFilename = "./bin/BlastData/salmonella-4k.txt";
//	std::string queryFilename = "./bin/BlastData/salmonella-6k.txt";
//	std::string queryFilename = "./bin/BlastData/salmonella-8k.txt";
//	std::string queryFilename = "./bin/BlastData/salmonella-10k.txt";
//	std::string queryFilename = "./bin/BlastData/salmonella-20k.txt";
//	std::string queryFilename = "./bin/BlastData/salmonella-30k.txt";
//	std::string queryFilename = "./bin/BlastData/salmonella-40k.txt";
//	std::string queryFilename = "./bin/BlastData/salmonella-50k.txt";
        
//	std::string dbFilename = "./bin/BlastData/ecoli-k12-231k-binary.txt";
//	std::string dbFilename = "./bin/BlastData/ecoli-k12-binary.txt";

	std::string db = "";
	std::string query = "";
	std::string tmp = "";

	std::ifstream queryFile;
        std::string queryFileName = QUERYFILENAME;
	queryFile.open(queryFileName.c_str(), std::ifstream::binary);

	std::ifstream dbFile;
        std::string dbFileName = DBFILENAME;
	dbFile.open(dbFileName.c_str(), std::ifstream::binary);

	queryFile.seekg(0, queryFile.end);
	int len = queryFile.tellg();
	char* cquery = new char[len];
	queryFile.seekg(0, queryFile.beg);
	queryFile.read(cquery, len);
	query = std::string(cquery, len);
	delete[] cquery;
	queryFile.close();

	dbFile.seekg(0, dbFile.end);
	len = dbFile.tellg();
	char* cdb = new char[len];
	dbFile.seekg(0, dbFile.beg);
	dbFile.read(cdb, len);
	db = std::string(cdb, len);
	delete[] cdb;
	dbFile.close();

	const int BUFFER_CAPACITY = len;

////////////////////////////////////
#if 1
  // print experiment params if desired
  // NB: all possible topos included for sanity check
  // convert topology indicators to string
#if RUN_BLASTAPP
  const char topoString[] = "BLAST";
#elif RUN_BLASTUBERAPP
  const char topoString[] = "BLASTUBER";
#elif RUN_BLAST2MODULESAPP
  const char topoString[] = "BLAST2Modules";
#else
  const char topoString[] = "NONSTANDARD";
#endif

      // convert mapping indicators to string
#if MAPPING_1TO1
      const char mapString[] = "1-to-1";
#elif MAPPING_1TO2
      const char mapString[] = "1-to-2";
#elif MAPPING_1TO4
      const char mapString[] = "1-to-4";
#elif MAPPING_2TO1
      const char mapString[] = "2-to-1";
#elif MAPPING_4TO1
      const char mapString[] = "4-to-1";
#else
      const char mapString[] = "NONSTANDARD";
#endif

      // print app metadata
      printf("APP PARAMS: TOPOLOGY: %s ELTS-TO-THREADS MAPPING: %s FILTER_RATE: %.2f INPUTS: %d\n", 
          topoString, mapString, FILTER_RATE, len); 

      //debug
//      printf("Size of input item: %d\n", sizeof(PipeEltT));
#endif
////////////////////////////////////

	//Construct CPU Query Hash Table
	node** queryHashes = new node*[HASH_SIZE];
	node* n;

	//node* a;
	for(int i = 0; i < HASH_SIZE; ++i) {
		//a = new node;
		//a->x = -1;
		//a->next = NULL;
		queryHashes[i] = NULL;
	}

	for(int i = 0; i < query.size() - 9; ++i) {
		//int word = ((int)((unsigned char)query.at(i))) * 256 + (unsigned char)query.at(i + 1);
		int word = 0;
		for(int j = 0; j < 8; ++j) {
			word = word << 2;
			char curr = query.at(i + j);
			switch(curr) {
				case 'G':
					//G
					word += 3;
					break;
				case 'T':
					//T
					word += 2;
					break;
				case 'C':
					//C
					word += 1;
					break;
				case 'A':
					//A
					word += 0;
					break;
				default:
					break;
			};
		}
		node* n2 = new node;
		n2->x = i;
		n2->next = queryHashes[word];
		queryHashes[word] = n2;
	}

	  int* inBufferData;
	  hipMallocManaged(&inBufferData, BUFFER_CAPACITY * sizeof(int));
      gpuErrchk( hipPeekAtLastError() );
      gpuErrchk( hipDeviceSynchronize() );

	  Mercator::InputBuffer<int>* inBuffer = new Mercator::InputBuffer<int>(inBufferData, BUFFER_CAPACITY);

//	  std::string sourceNodeName = "sourceNode";

	  // set up output buffers
//	  std::vector<Mercator::IO::BufferBase*> outBufferVec;
//	  std::vector<std::string>               sinkNodeNameVec;

	  Mercator::OutputBuffer<point>* outBuff = new Mercator::OutputBuffer<point>(BUFFER_CAPACITY);

//	  outBufferVec.push_back(outBuff);
//	  std::string sinkNodeName = "sinkNodeAccept";
//	  sinkNodeNameVec.push_back(sinkNodeName);

	  // fill input buffer
	  for(int i=0; i < BUFFER_CAPACITY; ++i)
	    inBuffer->add(i);

	// print contents of input buffer
	//for(int i=0; i < inBuffer->get_numItems(); ++i)
	 // std::cout << "Input[" << i << "]: " << inBuffer->peek(i) << "\n";

	  // create app object
//	  Mercator::App* blastApp = new Mercator::App();
          APP_TYPE* blastApp = new APP_TYPE();

		//Construct GPU Query Hash Table
		int* qHits = new int[HASH_SIZE];
		int* qHash = new int[HASH_SIZE];
		for(int i = 0; i < 2; ++i) {
			qHits[i] = -1;
		}

		std::vector<int> vec = std::vector<int>();
		int pos = 2;
		for(int i = 0; i < HASH_SIZE; ++i){
			n = queryHashes[i];
			if(!n) {
				//no hits
				qHash[i] = -1;
			}
			else if(!n->next) {
				//single hit
				qHash[i] = n->x;
				//std::cout << qHash[i] << " ";
			}
			else {				
				//multiple hits
				//n = n->next;
				while(n) {
					vec.push_back(n->x);
					n = n->next;
				}
				//do array stuffs
				int* newarr = new int[pos + vec.size() + 1];
				memcpy(newarr, qHits, sizeof(int) * pos);
				for(int j = 0; j < vec.size(); ++j) {
					newarr[pos + j] = vec.at(j);
					//if(vec.at(j) == 48421) {
					//	std::cout << "H " << i << std::endl;
					//	std::string rsp3;
					//	std::cin >> rsp3;
					//}
				}
				newarr[pos + vec.size()] = -1;
				delete[] qHits;
				qHits = newarr;
				qHash[i] = pos * -1;
				pos += vec.size() + 1;
				//std::cout << vec.size() << ": ";
				//for(int j = 0; j < vec.size(); ++j)
				//	std::cout << vec.at(j) << " ";
				//std::cout << std::endl;
				vec = std::vector<int>();
				//std::string rsp;
				//std::cin >> rsp;
			}
			//std::cout << i << ": " << qHash[i] << std::endl;
			//if(i % 1000 == 0) {
			//	std::string rsp;
			//	std::cin >> rsp;
			//}
		}
		//std::cout << qHash[0x2bcd] << std::endl;
		/*
		int p = 0;
		int pMax = 0;
		for(int i = 0; i < pos; ++i) {
			if(qHits[i] == -1) {
				std::cout << qHits[i] << std::endl;
				pMax = max(p, pMax);
				p = 0;
			}
			else {
				std::cout << qHits[i] << " ";
				++p;
			}
		}
		std::cout << "pMAX = " << pMax << std::endl;
		std::string rsp2;
		std::cin >> rsp2;
		*/
		
		//Allocate c strings for query and databse (to go to GPU)
		int qSize = query.size();
		int dSize = db.size();
		Base* q = (Base*)malloc((qSize + STRING_BUFF * 8) * sizeof(Base));
		Base* d = (Base*)malloc((dSize + STRING_BUFF * 2) * sizeof(Base));

		//Buffer beginning of c strings
		for(int i = 0; i < STRING_BUFF; ++i) {
			for(int j = 0; j < 4; ++j) {
				q[i] = 'A';
			}
			d[i] = 0xFF;
		}

		//Fill query c string
		for(int i = STRING_BUFF * 4; i < qSize + STRING_BUFF * 4; ++i) {
			q[i] = query.at(i - STRING_BUFF * 4);
		}

		//Fill database c string
		for(int i = STRING_BUFF; i < dSize + STRING_BUFF; ++i) {
			d[i] = db.at(i - STRING_BUFF);
		}

		//Buffer end of c strings
		for(int i = 0; i < STRING_BUFF; ++i) {
			for(int j = 0; j < 4; ++j) {
				q[i + qSize + STRING_BUFF * 4] = 'A';
			}
			d[i + dSize + STRING_BUFF] = 0xFF;
		}

		//Initialize global app data on GPU
		BlastData* blastAppData = new BlastData(qHits, qHash, pos, HASH_SIZE, q, d, qSize + STRING_BUFF * 8, dSize + STRING_BUFF * 2);
		blastApp->set_userData(blastAppData);


  // associate buffers with nodes
  blastApp->sourceNode->set_inBuffer(inBuffer);
  blastApp->sinkNodeAccept->set_outBuffer(outBuff);

// print contents of output buffer
std::cout << "Calling run fcn for blast app... \n" ;

  // run main function
  blastApp->run();

  printf("BlastUber app finished. Num results: %d\n", outBuff->size());

#if PRINT_OUTPUT_BUFFERS_BLAST
// print contents of output buffer
std::cout << "Output buffer: \n" ;

point* outData = outBuff->get_data();
for(int i=0; i < outBuff->size(); ++i)
   printf("[%d]: (%d, %d)\n", i, outData[i].db, outData[i].query);

#endif

hipFree(inBufferData);

// NB: pretty-printing currently broken
#if 0

//PRETTY PRINT

        // main pretty-print loop
        std::cout << "*** Formatted results.  Total results: " 
          << outBuff->size() << std::endl;
	for(int j = 0; j < outBuff->size(); ++j) {  
          auto nextPoint = outData[j];
	std::cout << "J = " << j << std::endl;
	int tmpScore = 0;
	int highestL = 0;
	int highestR = 0;
	int tmpr = 8;
	int tmpl = 0;
	int iMin = min(nextPoint.db, nextPoint.query / 4);
	int iMax = min(dSize - nextPoint.db, (qSize - nextPoint.query + 3) / 4);
	const Base* queryy = q + STRING_BUFF * 4 + nextPoint.query;
	const Base* dbb = d + STRING_BUFF + nextPoint.db;
	int mask;
	for(int i = 1; i <= iMin; ++i) {
		int qbyte = queryy[i * -4 + 3];
		int dbyte = dbb[-i];

		for (int k = 0; k < 4; ++k) {
			mask = 0x03 << (2 * k);
			tmpScore += (((qbyte & 0x06) >> 1) == ((dbyte & mask) >> (2 * k)) ? MATCH_SCORE : MISMATCH_SCORE);
			qbyte = queryy[i * -4 + 3 - k - 1];
			if(tmpScore > highestL) {
				highestL = tmpScore;
				tmpl = i - STRING_BUFF;
			}
		}
		if(highestL + tmpScore < MAX_DIFF) {
			break;
		}
	}
	tmpScore = 0;

	for(int i = 2; i <= iMax; ++i) {

		int qbyte = query[i * 4];
		int dbyte = db[i];
      
		for (int k = 0; k < 4; ++k) {
			mask = 0xC0 >> (2 * k);
			tmpScore += (((qbyte & 0x06) << 5) == ((dbyte & mask) << (2 * k)) ? MATCH_SCORE : MISMATCH_SCORE);
			qbyte = query[i * 4 + k + 1];
			if(tmpScore > highestL) {
				highestR = tmpScore;
				tmpr = i + 2 - STRING_BUFF;
			}
		}
		if(highestR + tmpScore < MAX_DIFF) {
			break;
		}
	}

	//Printing Matches
	std::string qTmp = query.substr(nextPoint.query - tmpl * 4, tmpl * 4 + 8 + tmpr * 4);
	std::string dTmp = "";
	//std::cout << "HE" << std::endl;
	/*
	for(int k = nextPoint.query - tmpl; k < nextPoint.query + tmpr; ++k) {
		assert(k > 0);
		assert(k <= query.size());
		char curr = query.at(k);
		switch(curr & 0xC0) {
			case 0xC0:
				//G
				qTmp += "G";
				break;
			case 0x80:
				//T
				qTmp += "T";
				break;
			case 0x40:
				//C
				qTmp += "C";
				break;
			case 0x00:
				//A
				qTmp += "A";
				break;
			default:
				qTmp += "X";
				break;
		};
		switch(curr & 0x30) {
			case 0x30:
				//G
				qTmp += "G";
				break;
			case 0x20:
				//T
				qTmp += "T";
				break;
			case 0x10:
				//C
				qTmp += "C";
				break;
			case 0x00:
				//A
				qTmp += "A";
				break;
			default:
				qTmp += "X";
				break;
		};
		switch(curr & 0x0C) {
			case 0x0C:
				//G
				qTmp += "G";
				break;
			case 0x08:
				//T
				qTmp += "T";
				break;
			case 0x04:
				//C
				qTmp += "C";
				break;
			case 0x00:
				//A
				qTmp += "A";
				break;
			default:
				qTmp += "X";
				break;
		};
		switch(curr & 0x03) {
			case 0x03:
				//G
				qTmp += "G";
				break;
			case 0x02:
				//T
				qTmp += "T";
				break;
			case 0x01:
				//C
				qTmp += "C";
				break;
			case 0x00:
				//A
				qTmp += "A";
				break;
			default:
				qTmp += "X";
				break;
		};
	}
	*/
	//qTmp = qTmp.substr((tmpl + 1) % 4 , qTmp.size() - ((tmpl + 1) / 4) - ((tmpr + 3) / 4));
	//qTmp = qTmp.substr(tmpl % 4, tmpr - tmpl + 9);
	//std::cout << qTmp << " " << qTmp.size() << std::endl;
	//std::string rsp;
	//std::cin >> rsp;
	for(int k = nextPoint.db - tmpl; k < nextPoint.db + tmpr + 2; ++k) {
		assert(k > 0);
		assert(k <= db.size());
		char curr = db.at(k);
		switch(curr & 0xC0) {
			case 0xC0:
				//G
				dTmp += "G";
				break;
			case 0x80:
				//T
				dTmp += "T";
				break;
			case 0x40:
				//C
				dTmp += "C";
				break;
			case 0x00:
				//A
				dTmp += "A";
				break;
			default:
				dTmp += "X";
				break;
		};
		switch(curr & 0x30) {
			case 0x30:
				//G
				dTmp += "G";
				break;
			case 0x20:
				//T
				dTmp += "T";
				break;
			case 0x10:
				//C
				dTmp += "C";
				break;
			case 0x00:
				//A
				dTmp += "A";
				break;
			default:
				dTmp += "X";
				break;
		};
		switch(curr & 0x0C) {
			case 0x0C:
				//G
				dTmp += "G";
				break;
			case 0x08:
				//T
				dTmp += "T";
				break;
			case 0x04:
				//C
				dTmp += "C";
				break;
			case 0x00:
				//A
				dTmp += "A";
				break;
			default:
				dTmp += "X";
				break;
		};
		switch(curr & 0x03) {
			case 0x03:
				//G
				dTmp += "G";
				break;
			case 0x02:
				//T
				dTmp += "T";
				break;
			case 0x01:
				//C
				dTmp += "C";
				break;
			case 0x00:
				//A
				dTmp += "A";
				break;
			default:
				dTmp += "X";
				break;
		};
	}
	//dTmp = dTmp.substr((tmpl + 1) % 4, dTmp.size() - ((tmpl + 1) / 4) - ((tmpr + 3) / 4));
	//dTmp = dTmp.substr(tmpl % 4, tmpr - tmpl + 9);
	//if(tmpl % 4 == 3)
	//	tmpl -= 2;
	//else if(tmpl % 4 == 1)
	//	tmpl += 2;
	std::cout << "DTMP = " << dTmp << "  " << dTmp.size() << std::endl;
	std::cout << "QTMP = " << qTmp << "  " << qTmp.size() << std::endl;
	std::cout << "--------------------------------------------" << std::endl;
	std::cout << nextPoint.query - tmpl << "\t" << std::hex << nextPoint.query - tmpl << std::dec << "\t" << qTmp << "\t" << nextPoint.query + tmpr << std::hex << "\t" << nextPoint.query + tmpr << std::dec << std::endl << "\t\t";
	for(int h = 0; h < qTmp.size(); ++h) {
		//assert(qTmp.size() == dTmp.size());
		//std::cout << "HERE";
		std::cout << (qTmp.at(h) == dTmp.at(h) ? "|" : " ");
	}
	std::cout << std::endl << nextPoint.db * 4 - tmpl * 4 << "\t" << std::hex << nextPoint.db - tmpl << std::dec << "\t" << dTmp << "\t" << nextPoint.db * 4 + tmpr * 4 << std::hex << "\t" << nextPoint.db + tmpr << std::dec << std::endl;
	std::cout << std::dec;
	} // end main pretty-print loop

#endif

// cleanup
hipFree(blastApp);

}

#endif
