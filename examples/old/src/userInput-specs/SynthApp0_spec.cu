
#include <hip/hip_runtime.h>
/**
  * Test input file for MERCATOR.
  * Tests a multistage pipeline with multiple sinks,
  *  AND multiple module types.
  * NB: this app is an extension of App0 with multiple 
  *      module types.
  * Thread/item mapping is 1/1 for all modules.
  * Topology: Source -> A -> B -> Sink1
  *           A -> Sink2
  *           B -> Sink3
  */

/*** App name. ***/
#pragma mtr application SynthApp0

/*** Module (i.e., module type) specs. ***/

// Filter1
#pragma mtr module Filter1<MyModuleData> (int[32] -> accept<int>:4, reject<int>:4 | 1 : 1) 

// Filter2
#pragma mtr module Filter2<MyModuleData> (int[32] -> accept<int>:4, reject<int>:4 | 1 : 1) 

// SOURCE Module
#pragma mtr module SOURCE<int>

// SINK Module
#pragma mtr module SINK<int>

/*** Node (i.e., module instance) specs. ***/
#pragma mtr node sourceNode : SOURCE
#pragma mtr node filter1node<MyNodeData> : Filter1
#pragma mtr node filter2node<MyNodeData> : Filter2
#pragma mtr node sinkNodeReject1 : SINK<int>
#pragma mtr node sinkNodeReject2 : SINK<int>
#pragma mtr node sinkNodeAccept : SINK<int>


/*** Edge specs. ***/

// SOURCE -> Filter1
#pragma mtr edge sourceNode::outStream -> filter1node

// Filter1 -> Filter2
#pragma mtr edge filter1node::accept -> filter2node

// Filter1 -> Rejects
#pragma mtr edge filter1node::reject -> sinkNodeReject1

// Filter2 -> SINK
#pragma mtr edge filter2node::accept -> sinkNodeAccept

// Filter2 -> Rejects
#pragma mtr edge filter2node::reject -> sinkNodeReject2
